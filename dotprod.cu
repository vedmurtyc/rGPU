
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

#define SIZE 1000
#define NUM_BLOCKS 10
#define THREADS_PER_BLOCK 100

__global__ void DotProd(int *a, int *b, int *c) {
	
	__shared__ int temp[THREADS_PER_BLOCK];

	int x = threadIdx.x + blockDim.x * blockIdx.x;
	/*printf("Block ID :%d:\n", blockIdx.x);
	printf("Block Dim :%d:\n", blockDim.x);
	printf("Theard ID :%d:\n", threadIdx.x);*/
	temp[threadIdx.x] = a[x] * b[x];
	// printf("Temp:%d\n", temp[threadIdx.x]);

	__syncthreads();
	
	if (threadIdx.x == 0) 
	{
		int i,sum = 0;
		for (i = 0; i < THREADS_PER_BLOCK; i++) 
		{
			sum += temp[i];
		}
		// printf("\nSUM[%d]:%d", blockIdx.x, sum);
		atomicAdd(c, sum);
	}
}


int main() {
	int *a, *b, *c;
	int *d_a, *d_b, *d_c;
	int n = SIZE * sizeof(int);
	int i;
	
	// STEP 1 : Allocate memory for Host and Device variables
	a = (int*)malloc(n);
	b = (int*)malloc(n);
	c = (int*)malloc(sizeof(int));
	
	hipMalloc(&d_a, n);
	hipMalloc(&d_b, n);
	hipMalloc(&d_c, sizeof(int));

	// STEP 2: Initialize Host variables
	*c = 0;
	for (i = 0; i < SIZE; i++) {
		a[i] = i + 1;
		b[i] = 2 * (i + 1);
	}
	
	// Display the values of the arrays
	printf("\nArray A:\n");
	for (i = 0; i < SIZE; i++) {
		printf("%d ", a[i]);
	}
	printf("\nArray B:\n");
	for (i = 0; i < SIZE; i++) {
			printf("%d ", b[i]);
	}
	printf("\n");
	// STEP 3: Copy data to device variables.
	hipMemcpy(d_a, a, n, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, n, hipMemcpyHostToDevice);
	hipMemcpy(d_c, c, sizeof(int), hipMemcpyHostToDevice);

	// STEP 4: Launch the Kernel
	printf("\nLaunching Kernel\n");
	DotProd <<<NUM_BLOCKS, THREADS_PER_BLOCK>>> (d_a, d_b, d_c);

	//STEP 5: Copy results from device to Host.
	hipMemcpy(c, d_c, sizeof(int), hipMemcpyDeviceToHost);
	
	printf("\nDot Product is: %d\n", *c);
	
	//STEP 6: Free Memory
	free(a); free(b); free(c);
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	
	return 0;
}
